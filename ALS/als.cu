#include "hip/hip_runtime.h"

#define SURPASS_NAN
// #define USE_CG
//if cojugate gradient solver generates results in FP16
//#define CUMF_TT_FP16
//#define CUMF_XX_FP16
#define CG_ITER 6
//#define CUMF_SAVE_MODEL
#include "als.h"
//#include "device_utilities.h"
#include "host_utilities.h"
#include <fstream>
#include <assert.h>
#include<stdlib.h>
#include<stdio.h>
#include <string>
#define SCAN_BATCH 28
#include <iostream>
using namespace std;

void saveDeviceFloatArrayToFile(string fileName, int size, float* d_array){
	float* h_array;
	cudacall(hipHostMalloc( (void** ) &h_array, size * sizeof(h_array[0])) );
	cudacall(hipMemcpy(h_array, d_array, size * sizeof(h_array[0]),hipMemcpyDeviceToHost));
	FILE * outfile = fopen(fileName.c_str(), "wb");
	fwrite(h_array, sizeof(float), size, outfile);
	fclose(outfile);
	hipHostFree(h_array);
}
int updateX(const int batch_size, const int batch_offset, float * ythetaT, float * tt, float * XT,
		hipblasHandle_t handle, const int m, const int n, const int f, const int nnz,
		float** devPtrTTHost, float **devPtrYthetaTHost)
    {
      float **devPtrTT = 0;
    	int *INFO;
    	for (int k = 0; k < batch_size; k++) {
    		devPtrTTHost[k] = &tt[k * f * f];
    	}
    	cudacall(hipMalloc((void** ) &devPtrTT, batch_size * sizeof(*devPtrTT)));
    	cudacall(hipMemcpy(devPtrTT, devPtrTTHost, batch_size * sizeof(*devPtrTT),hipMemcpyHostToDevice));
    	//cudacall( hipMalloc(&P, f * batch_size * sizeof(int)) );
    	cudacall( hipMalloc(&INFO, batch_size * sizeof(int) ));

      //Performing matrix inversion
    	cublascall(hipblasSgetrfBatched(handle, f, devPtrTT, f, NULL, INFO, batch_size));

    	hipDeviceSynchronize();

      float **devPtrYthetaT = 0;

    	for (int k = 0; k < batch_size; k++) {
    		devPtrYthetaTHost[k] = &ythetaT[batch_offset * f + k * f];
    	}
    	cudacall(hipMalloc((void** ) &devPtrYthetaT, batch_size * sizeof(*devPtrYthetaT)));
    	cudacall(hipMemcpy(devPtrYthetaT, devPtrYthetaTHost, batch_size * sizeof(*devPtrYthetaT), hipMemcpyHostToDevice));

    	int * info2 = (int *) malloc(sizeof(int));
    	cublascall( hipblasSgetrsBatched(handle, HIPBLAS_OP_N, f, 1,
    			(const float ** ) devPtrTT, f, NULL, devPtrYthetaT, f, info2, batch_size) );

    	hipDeviceSynchronize();
    	hipError_t cudaStat1 = hipGetLastError();
    	if (cudaStat1 != hipSuccess) {
    		fprintf(stderr,"Failed to launch hipblasSgetrsBatched (error code: %s)!\n", hipGetErrorString(cudaStat1));
    		exit(EXIT_FAILURE);
    	}

    	cudacall( hipMemcpy(&XT[batch_offset * f], &ythetaT[batch_offset * f],
    			batch_size * f * sizeof(float), hipMemcpyDeviceToDevice) );

      cudacall(hipFree(devPtrTT));
    	//cudacall(hipFree(P));
    	cudacall(hipFree(INFO));
    	cudacall(hipFree(devPtrYthetaT));
    	return 0;

    }

		int updateTheta(const int batch_size, const int batch_offset, float * xx, float * yTXT, float * thetaT,
				hipblasHandle_t handle, const int m, const int n, const int f, const int nnz,float ** devPtrXXHost, float **devPtrYTXTHost )
		    {
		      float **devPtrXX = 0;

		    	for (int k = 0; k < batch_size; k++) {
		    		devPtrXXHost[k] = &xx[k * f * f];
		    	}
		    	cudacall(hipMalloc((void** ) &devPtrXX, batch_size * sizeof(*devPtrXX)));
		    	cudacall(hipMemcpy(devPtrXX, devPtrXXHost, batch_size * sizeof(*devPtrXX), hipMemcpyHostToDevice));

		      int *INFO;

		    	cudacall(hipMalloc(&INFO, batch_size * sizeof(int)));
		    	cublascall(hipblasSgetrfBatched(handle, f, devPtrXX, f, NULL, INFO, batch_size));
		    	hipDeviceSynchronize();

		      float **devPtrYTXT = 0;

		    	for (int k = 0; k < batch_size; k++) {
		    		devPtrYTXTHost[k] = &yTXT[batch_offset * f + k * f];
		    	}

		      cudacall(hipMalloc((void** ) &devPtrYTXT, batch_size * sizeof(*devPtrYTXT)));
		    	cudacall(hipMemcpy(devPtrYTXT, devPtrYTXTHost, batch_size * sizeof(*devPtrYTXT),hipMemcpyHostToDevice));

		    	int * info2 = (int *) malloc(sizeof(int));
		    	cublascall( hipblasSgetrsBatched(handle, HIPBLAS_OP_N, f, 1,
		    			(const float ** ) devPtrXX, f, NULL, devPtrYTXT, f, info2, batch_size) );
		    	hipDeviceSynchronize();
		    	hipError_t cudaStat1 = hipGetLastError();
		    	if (cudaStat1 != hipSuccess) {
		    		fprintf(stderr,"Failed to launch hipblasSgetrsBatched (error code: %s)!\n", hipGetErrorString(cudaStat1));
		    		exit(EXIT_FAILURE);
		    	}

		    	cudacall( hipMemcpy( &thetaT[batch_offset * f], &yTXT[batch_offset * f],
		    	                        batch_size * f * sizeof(float), hipMemcpyDeviceToDevice) );

		      hipFree(devPtrXX);
		    	hipFree(INFO);
		    	free(info2);
		    	hipFree(devPtrYTXT);
		    	return 0;

		    }

				__global__
				void RMSE(const float * csrVal, const int* cooRowIndex, const int* csrColIndex, const float * __restrict__ thetaT,
				    const float * __restrict__ XT, float * error, const int nnz, const int error_size, const int f)
				    {
				      int i = blockIdx.x * blockDim.x + threadIdx.x;
				      if(i<nnz)
				      {
				        int row = cooRowIndex[i];
				        int col = csrColIndex[i];
				        float e = csrVal[i];

				        for(int k=0;k<f;k++)
				        {
				          #ifdef SURPASS_NAN
				    			//a and b could be; there are user/item in testing but not training set
				    			float a = thetaT[f * col + k];
				    			float b = XT[f * row + k];
				    			//if(isnan(a)||isnan(b))//nan not working in some platform
				    			if(a!=a||b!=b)
				    				break;
				    			else
				    				e -= a * b;
				    			//if(isnan(a)) printf("row: %d, col: %d\n", row, col);
				    			//if(isnan(b)) printf("b[%d]: %f.\n", i, b);
				    			#else
				    			e -= thetaT[f * col + k] * XT[f * row + k];
				    			#endif
				        }
				        atomicAdd(&error[i%error_size], e*e);
				      }
				    }



/*a generic kernel to get the hermitian matrices
 * as the left-hand side of the equations, to update X in ALS
 *examplary F = 100, T = 10
 */
 __global__
 void get_hermitianT10(const int batch_offset, float *tt,
                     const int *csrRowIndex, const int *csrColIndex, const float lambda,
                   const int m, const int F, const float * __restrict__ thetaT)
 {
   extern __shared__ float2 thetaTemp[];
   // Each row of Rating has a thread block and the offset is the Number of rows aldready computed
   int row = blockIdx.x + batch_offset;
   if(row < m)
   {
     int start  = csrRowIndex[row];
     int end = csrRowIndex[row+1];

     // Number of non zero elements in the ith row of the Rating Matrix
     //SCAN_BATCH is probalbly the bin size ie. to be loaded in the shared memeory
     int iterations = (end-start+1)/SCAN_BATCH + 1;

     float temp0= 0, temp1= 0, temp2= 0, temp3= 0, temp4= 0, temp5= 0, temp6= 0, temp7= 0, temp8= 0, temp9 = 0;
 		float temp10= 0, temp11= 0, temp12= 0, temp13= 0, temp14= 0, temp15= 0, temp16= 0, temp17= 0, temp18= 0, temp19 = 0;
 		float temp20= 0, temp21= 0, temp22= 0, temp23= 0, temp24= 0, temp25= 0, temp26= 0, temp27= 0, temp28= 0, temp29 = 0;
 		float temp30= 0, temp31= 0, temp32= 0, temp33= 0, temp34= 0, temp35= 0, temp36= 0, temp37= 0, temp38= 0, temp39 = 0;
 		float temp40= 0, temp41= 0, temp42= 0, temp43= 0, temp44= 0, temp45= 0, temp46= 0, temp47= 0, temp48= 0, temp49 = 0;
 		float temp50= 0, temp51= 0, temp52= 0, temp53= 0, temp54= 0, temp55= 0, temp56= 0, temp57= 0, temp58= 0, temp59 = 0;
 		float temp60= 0, temp61= 0, temp62= 0, temp63= 0, temp64= 0, temp65= 0, temp66= 0, temp67= 0, temp68= 0, temp69 = 0;
 		float temp70= 0, temp71= 0, temp72= 0, temp73= 0, temp74= 0, temp75= 0, temp76= 0, temp77= 0, temp78= 0, temp79 = 0;
 		float temp80= 0, temp81= 0, temp82= 0, temp83= 0, temp84= 0, temp85= 0, temp86= 0, temp87= 0, temp88= 0, temp89 = 0;
 		float temp90= 0, temp91= 0, temp92= 0, temp93= 0, temp94= 0, temp95= 0, temp96= 0, temp97= 0, temp98= 0, temp99 = 0;

     int N = F/T10;
     int effective_block_size = N*(N+1)/2;

     int tile_x = 0;
     int tile_y = 0;

     for(int i=0;i<N;i++)
     {
       int end = ((2*N-1)*(i+1))/2;
       if(threadIdx.x<end)
       {
         tile_x = i*T10;
         tile_y = (N + threadIdx.x - end) * T10;
         break;
       }
     }

     int index = blockIdx.x*F*F;


     for(int iter=0;iter<iterations;iter++)
     {
       //Phase 1: Copying from Global Memory to shared Memory
       if(threadIdx.x<F/2)
       {
         for(int k=0;k<SCAN_BATCH;k++)
         {
           if(iter*SCAN_BATCH+k<end-start) //Border Condition
           {
             float2 theta;
             //Stored in column majour order
 						theta.x = thetaT[F * csrColIndex[start + iter*SCAN_BATCH + k] + 2*threadIdx.x];
 						theta.y = thetaT[F * csrColIndex[start + iter*SCAN_BATCH + k] + 2*threadIdx.x+1];
 						thetaTemp[k * F/2 + threadIdx.x] = theta;
           }
           //not enough theta to copy, set zero
 					else
 						memset(&thetaTemp[k*F/2 + threadIdx.x], 0, 2*sizeof(float));
         }
       }

       __syncthreads();

       //Phase 2: calculating A and storing in register
       //tid = 0 calculates the first 10 in thetaTemp and thetaTempTrans, tid = 1 calculates first 10 in thetaTemp and 10-20 in thetaTempTrans...
       // so total threads needed is 10(for the first 10 elements in thetaTemp with every set of 10 elements in thetaTempTrans)+
       // 9(second 10 elements in thetaTemp with set 1-9 elements thetaTempTrans)
       // + 8 + .. + 1 = 55 we are not using 100 threads as there are multilpe duplicate calculations
       if(threadIdx.x < effective_block_size)
       {
         for(int k = 0; k < SCAN_BATCH; k++){
 					accumulate_in_registers();
 				}
       }
     }
     //end of iteration in copying from smem and aggregating in register
 		__syncthreads();

     //Phase 3: Copying from registers to global memory
     		if(threadIdx.x < effective_block_size)
         {
           fill_lower_half_from_registers();

           if(tile_x != tile_y)
           {
 				        fill_upper_half_from_registers();
 			    }

           if(tile_x == tile_y)
           {
 				        for(int k = 0; k < T10; k++)
 					           tt[index + (tile_x+k)*(1+F)] += (end - start) * lambda;
 			    }
         }
   }
 }

float doALS(const int* csrRowIndexHostPtr, const int* csrColIndexHostPtr, const float* csrValHostPtr,
		const int* cscRowIndexHostPtr, const int* cscColIndexHostPtr, const float* cscValHostPtr,
		const int* cooRowIndexHostPtr, float* thetaTHost, float* XTHost,
		const int * cooRowIndexTestHostPtr, const int * cooColIndexTestHostPtr, const float * cooValHostTestPtr,
		const int m, const int n, const int f, const long nnz, const long nnz_test, const float lambda,
		const int ITERS, const int X_BATCH, const int THETA_BATCH, const int DEVICEID)
{
	hipSetDevice(DEVICEID);
	printf("*******parameters: m: %d, n:  %d, f: %d, nnz: %ld \n", m, n, f, nnz);
	//Creating device pointers
	int * csrRowIndex = 0;
	int * csrColIndex = 0;
	float * csrVal = 0;
	float * thetaT = 0;
	float * tt = 0;
	float * XT = 0;
	float * cscVal =0;
	int * cscRowIndex = 0;
	int * cscColIndex = 0;

	//Coo format is used for calculating the root mean square error
	int * cooRowIndex =0;
	float * cooVal_test;
	int * cooRowIndex_test;
	int * cooColIndex_test;
	float final_rmse = 0;
	//Allocating memeory to the device pointers

	cudacall(hipMalloc((void** ) &cscRowIndex,nnz * sizeof(cscRowIndex[0])));
	cudacall(hipMalloc((void** ) &cscColIndex, (n+1) * sizeof(cscColIndex[0])));
	cudacall(hipMalloc((void** ) &cscVal, nnz * sizeof(cscVal[0])));

	//thetaT : f * N
	cudacall(hipMalloc((void** ) &thetaT, f * n * sizeof(thetaT[0])));

	//X : M * f
	cudacall(hipMalloc((void** ) &XT, f * m * sizeof(XT[0])));

	//Copying data from host to device

	cudacall(hipMemcpy(cscRowIndex, cscRowIndexHostPtr,(size_t ) nnz * sizeof(cscRowIndex[0]), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(cscColIndex, cscColIndexHostPtr,(size_t ) (n+1) * sizeof(cscColIndex[0]), hipMemcpyHostToDevice));
	cudacall(hipMemcpy(cscVal, cscValHostPtr,(size_t ) (nnz * sizeof(cscVal[0])),hipMemcpyHostToDevice));
	cudacall(hipMemcpy(thetaT, thetaTHost, (size_t ) (n * f * sizeof(thetaT[0])), hipMemcpyHostToDevice));
	//CG needs XT
	cudacall(hipMemcpy(XT, XTHost, (size_t ) (m * f * sizeof(XT[0])), hipMemcpyHostToDevice));

	cudacall(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
	//To minimize bank conflics the size of the bank has been set to eight bytes
	//64-bit smem access
	//http://acceleware.com/blog/maximizing-shared-memory-bandwidth-nvidia-kepler-gpus
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

	//initialize cublas, cusparse
	hipblasHandle_t handle;
	cublascall(hipblasCreate(&handle));
	hipsparseHandle_t cushandle = 0;
	cusparsecall(hipsparseCreate(&cushandle));
	hipsparseMatDescr_t descr;
	cusparsecall( hipsparseCreateMatDescr(&descr));
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	for(int iter = 0; iter < ITERS ; iter ++){

		//copy csr matrix in
		cudacall(hipMalloc((void** ) &csrRowIndex,(m + 1) * sizeof(csrRowIndex[0])));
		cudacall(hipMalloc((void** ) &csrColIndex, nnz * sizeof(csrColIndex[0])));
		cudacall(hipMalloc((void** ) &csrVal, nnz * sizeof(csrVal[0])));
		cudacall(hipMemcpy(csrRowIndex, csrRowIndexHostPtr,(size_t ) ((m + 1) * sizeof(csrRowIndex[0])), hipMemcpyHostToDevice));
		cudacall(hipMemcpy(csrColIndex, csrColIndexHostPtr,(size_t ) (nnz * sizeof(csrColIndex[0])), hipMemcpyHostToDevice));
		cudacall(hipMemcpy(csrVal, csrValHostPtr,(size_t ) (nnz * sizeof(csrVal[0])),hipMemcpyHostToDevice));


		float * ytheta = 0;
		float * ythetaT = 0;
		cudacall(hipMalloc((void** ) &ytheta, f * m * sizeof(ytheta[0])));
		cudacall(hipMalloc((void** ) &ythetaT, f * m * sizeof(ythetaT[0])));

		const float alpha = 1.0f;
		const float beta = 0.0f;
		//hipsparseScsrmm2 give α ∗ op ( A ) ∗ op ( B ) + β ∗ C where A is a sparce matrix B and C are dense matrices
		cusparsecall (hipsparseScsrmm2(cushandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				HIPSPARSE_OPERATION_TRANSPOSE, m, f, n, nnz, &alpha, descr, csrVal,
				csrRowIndex, csrColIndex, thetaT, f, &beta, ytheta, m) );

		//printf("*******transpose ytheta use cublas.\n");
		//ytheta: m*f; need ythetaT = (ytheta).T = f*m
		//Summing up X*ThetaR over all iterations and storing in ythetaT
		cublascall(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, f, m, &alpha,
				(const float * ) ytheta, m, &beta, ythetaT, f, ythetaT, f));
		//hipDeviceSynchronize();
		//cudaCheckError();
		cudacall(hipFree(ytheta));
		cudacall(hipFree(csrVal));


		int block_dim = f/T10*(f/T10+1)/2;
		   //minimum number of threads is f/2 to load each column of ThetaTs
		if (block_dim < f/2) block_dim = f/2;
		for(int batch_id = 0; batch_id< X_BATCH; batch_id ++){

			int batch_size = 0;
			if(batch_id != X_BATCH - 1)
				batch_size = m/X_BATCH;
			else
				batch_size = m - batch_id*(m/X_BATCH);
			int batch_offset = batch_id * (m/X_BATCH);

			cudacall(hipMalloc((void** ) &tt, f * f * batch_size * sizeof(float)));

			//updateXByBlock kernel.

				get_hermitianT10<<<batch_size, block_dim, SCAN_BATCH * f/2*sizeof(float2)>>>
					(batch_offset, tt, csrRowIndex, csrColIndex, lambda, m, f, thetaT);
			hipDeviceSynchronize();
			cudaCheckError();


			float ** devPtrTTHost = 0;
			cudacall(hipHostMalloc( (void** ) &devPtrTTHost, batch_size * sizeof(*devPtrTTHost) ) );
			float **devPtrYthetaTHost = 0;
			cudacall(hipHostMalloc( (void** ) &devPtrYthetaTHost, batch_size * sizeof(*devPtrYthetaTHost) ) );
			updateX(batch_size, batch_offset, ythetaT, tt, XT, handle, m, n, f, nnz, devPtrTTHost, devPtrYthetaTHost);
			cudacall(hipHostFree(devPtrTTHost));
			cudacall(hipHostFree(devPtrYthetaTHost));

			cudacall(hipFree(tt));
		}

		cudacall(hipFree(csrRowIndex));
		cudacall(hipFree(csrColIndex));
		cudacall(hipFree(ythetaT));

///*
// ________UPDATE THETA_

		float * yTX = 0;
		float * yTXT = 0;
		cudacall(hipMalloc((void** ) &yTXT, f * n * sizeof(yTXT[0])));
		cudacall(hipMalloc((void** ) &yTX, n * f * sizeof(yTX[0])));
		cusparsecall( hipsparseScsrmm2(cushandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				HIPSPARSE_OPERATION_TRANSPOSE, n, f, m, nnz, &alpha, descr, cscVal,
				cscColIndex, cscRowIndex, XT, f, &beta, yTX, n) );
		//hipDeviceSynchronize();
		//printf("*******transpose yTX \n");
		//yTX: n*f; need yTXT = (yTX).T = f*n
		cublascall(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, f, n, &alpha,
				(const float * ) yTX, n, &beta, yTXT, f, yTXT, f));
		hipDeviceSynchronize();
		cudacall(hipFree(yTX));

		//in batches, when N is huge
		for(int batch_id = 0; batch_id< THETA_BATCH; batch_id ++){

			int batch_size = 0;
			if(batch_id != THETA_BATCH - 1)
				batch_size = n/THETA_BATCH;
			else
				batch_size = n - batch_id*(n/THETA_BATCH);
			int batch_offset = batch_id * (n/THETA_BATCH);

			float * xx = 0;

			cudacall(hipMalloc((void** ) &xx, f * f * batch_size * sizeof(xx[0])));
			cudacall( hipMemset(xx, 0, f*f*batch_size*sizeof(float)) );
			#
			//get_hermitian_theta<<<batch_size, 64>>>(batch_offset, xx, cscRowIndex, cscColIndex, lambda, n);
			//updateThetaByBlock2pRegDsmemTile<<<batch_size, F>>>

				get_hermitianT10<<<batch_size, block_dim, SCAN_BATCH*f*sizeof(float)>>>
					(batch_offset, xx, cscColIndex, cscRowIndex, lambda, n, f, XT);
			hipDeviceSynchronize();
			cudaCheckError();


			float ** devPtrXXHost = 0;
			cudacall(hipHostMalloc( (void** ) &devPtrXXHost, batch_size * sizeof(*devPtrXXHost) ) );
			float **devPtrYTXTHost = 0;
			cudacall(hipHostMalloc( (void** ) &devPtrYTXTHost, batch_size * sizeof(*devPtrYTXTHost) ) );
			updateTheta(batch_size, batch_offset, xx, yTXT, thetaT, handle, m,  n,  f,  nnz,
					devPtrXXHost, devPtrYTXTHost);


			cudacall(hipHostFree(devPtrXXHost));
			cudacall(hipHostFree(devPtrYTXTHost));

			cudacall(hipFree(xx));
		}
		cudacall(hipFree(yTXT));

		float * errors_train = 0;
		int error_size = 1000;
		cudacall(hipMalloc((void** ) &errors_train, error_size * sizeof(errors_train[0])));
		cudacall( hipMemset(errors_train, 0, error_size*sizeof(float)) );

		cudacall(hipMalloc((void** ) &cooRowIndex, nnz * sizeof(cooRowIndex[0])));
		cudacall(hipMemcpy(cooRowIndex, cooRowIndexHostPtr,(size_t ) (nnz * sizeof(cooRowIndex[0])), hipMemcpyHostToDevice));
		cudacall(hipMalloc((void** ) &csrColIndex, nnz * sizeof(csrColIndex[0])));
		cudacall(hipMalloc((void** ) &csrVal, nnz * sizeof(csrVal[0])));
		cudacall(hipMemcpy(csrColIndex, csrColIndexHostPtr,(size_t ) (nnz * sizeof(csrColIndex[0])), hipMemcpyHostToDevice));
		cudacall(hipMemcpy(csrVal, csrValHostPtr,(size_t ) (nnz * sizeof(csrVal[0])),hipMemcpyHostToDevice));

		RMSE<<<(nnz-1)/256 + 1, 256>>>
				(csrVal, cooRowIndex, csrColIndex, thetaT, XT, errors_train, nnz, error_size, f);
		hipDeviceSynchronize();
		cudaCheckError();
		cudacall(hipFree(cooRowIndex));
		cudacall(hipFree(csrColIndex));
		cudacall(hipFree(csrVal));

		float* rmse_train = (float*) malloc (sizeof(float));
		cublascall( hipblasSasum(handle, error_size, errors_train, 1, rmse_train) );

		hipDeviceSynchronize();
		printf("--------- Train RMSE in iter %d: %f\n", iter, sqrt((*rmse_train)/nnz));
		cudacall(hipFree(errors_train));


		float * errors_test = 0;
		cudacall(hipMalloc((void** ) &errors_test, error_size * sizeof(errors_test[0])));
		cudacall( hipMemset(errors_test, 0, error_size*sizeof(float)) );

		cudacall(hipMalloc((void** ) &cooRowIndex_test, nnz_test * sizeof(cooRowIndex_test[0])));
		cudacall(hipMemcpy(cooRowIndex_test, cooRowIndexTestHostPtr,(size_t ) (nnz_test * sizeof(cooRowIndex_test[0])), hipMemcpyHostToDevice));
		cudacall(hipMalloc((void** ) &cooColIndex_test, nnz_test * sizeof(cooColIndex_test[0])));
		cudacall(hipMalloc((void** ) &cooVal_test, nnz_test * sizeof(cooVal_test[0])));
		cudacall(hipMemcpy(cooColIndex_test, cooColIndexTestHostPtr,(size_t ) (nnz_test * sizeof(cooColIndex_test[0])), hipMemcpyHostToDevice));
		cudacall(hipMemcpy(cooVal_test, cooValHostTestPtr,(size_t ) (nnz_test * sizeof(cooVal_test[0])),hipMemcpyHostToDevice));

		RMSE<<<(nnz_test-1)/256, 256>>>(cooVal_test, cooRowIndex_test, cooColIndex_test, thetaT, XT,
				errors_test, nnz_test, error_size, f);
		hipDeviceSynchronize();
		cudaCheckError();

		cudacall(hipFree(cooRowIndex_test));
		cudacall(hipFree(cooColIndex_test));
		cudacall(hipFree(cooVal_test));

		float* rmse_test = (float*) malloc (sizeof(float));
		cublascall( hipblasSasum(handle, error_size, errors_test, 1, rmse_test) );
		hipDeviceSynchronize();
		final_rmse = sqrt((*rmse_test)/nnz_test);
		printf("--------- Test RMSE in iter %d: %f\n", iter, final_rmse);
		cudacall(hipFree(errors_test));
//*/
	}
	//copy feature vectors back to host
	cudacall(hipMemcpy(thetaTHost, thetaT, (size_t ) (n * f * sizeof(thetaT[0])), hipMemcpyDeviceToHost));
	cudacall(hipMemcpy(XTHost, XT, (size_t ) (m * f * sizeof(XT[0])), hipMemcpyDeviceToHost));
	cudacall(hipFree(thetaT));
	cudacall(hipFree(XT));
	cudacall(hipFree(cscVal));
	cudacall(hipFree(cscColIndex));
	cudacall(hipFree(cscRowIndex));
	//WARN: do not call hipDeviceReset inside ALS()
	//because the caller needs to access XT and thetaT which was in the same context
	//cudacall(hipDeviceReset());
	return final_rmse;
}

int main(int argc, char **argv)
{

  if(argc != 10){
		printf("Usage: give M, N, F, NNZ, NNZ_TEST, lambda, X_BATCH, THETA_BATCH and DATA_DIR.\n");
		printf("E.g., for netflix data set, use: \n");
		printf("./main 17770 480189 100 99072112 1408395 0.048 1 3 ./data/netflix/ \n");
		printf("E.g., for movielens 10M data set, use: \n");
		printf("./main 71567 65133 100 9000048 1000006 0.05 1 1 ./data/ml10M/ \n");
		return 0;
	}

  int f = atoi(argv[3]);
	if(f%T10!=0){
		printf("F has to be a multiple of %d \n", T10);
		return 0;
  }
    int m = atoi(argv[1]);
  	int n = atoi(argv[2]);
  	long nnz = atoi(argv[4]);
  	long nnz_test = atoi(argv[5]);
  	float lambda = atof(argv[6]);
  	int X_BATCH = atoi(argv[7]);
  	int THETA_BATCH = atoi(argv[8]);
  	std::string DATA_DIR(argv[9]);
  	printf("M = %d, N = %d, F = %d, NNZ = %ld, NNZ_TEST = %ld, lambda = %f\nX_BATCH = %d, THETA_BATCH = %d\nDATA_DIR = %s \n",
  			m, n, f, nnz, nnz_test, lambda, X_BATCH, THETA_BATCH, DATA_DIR.c_str());

    int ITERS = 1000;

    int* csrRowIndexHostPtr;
  	cudacall(hipHostMalloc( (void** ) &csrRowIndexHostPtr, (m + 1) * sizeof(csrRowIndexHostPtr[0])) );
  	int* csrColIndexHostPtr;
  	cudacall(hipHostMalloc( (void** ) &csrColIndexHostPtr, nnz * sizeof(csrColIndexHostPtr[0])) );
  	float* csrValHostPtr;
  	cudacall(hipHostMalloc( (void** ) &csrValHostPtr, nnz * sizeof(csrValHostPtr[0])) );
  	float* cscValHostPtr;
  	cudacall(hipHostMalloc( (void** ) &cscValHostPtr, nnz * sizeof(cscValHostPtr[0])) );
  	int* cscRowIndexHostPtr;
  	cudacall(hipHostMalloc( (void** ) &cscRowIndexHostPtr, nnz * sizeof(cscRowIndexHostPtr[0])) );
  	int* cscColIndexHostPtr;
  	cudacall(hipHostMalloc( (void** ) &cscColIndexHostPtr, (n+1) * sizeof(cscColIndexHostPtr[0])) );
  	int* cooRowIndexHostPtr;
  	cudacall(hipHostMalloc( (void** ) &cooRowIndexHostPtr, nnz * sizeof(cooRowIndexHostPtr[0])) );

    //calculate X from thetaT first, need to initialize thetaT
	float* thetaTHost;
	cudacall(hipHostMalloc( (void** ) &thetaTHost, n * f * sizeof(thetaTHost[0])) );

	float* XTHost;
	cudacall(hipHostMalloc( (void** ) &XTHost, m * f * sizeof(XTHost[0])) );

	//initialize thetaT on host
	unsigned int seed = 0;
	srand (seed);
	for (int k = 0; k < n * f; k++)
		thetaTHost[k] = 0.2*((float) rand() / (float)RAND_MAX);

  printf("*******start loading training and testing sets to host.\n");


  int* cooRowIndexTestHostPtr = (int *) malloc(nnz_test * sizeof(cooRowIndexTestHostPtr[0]));

  int* cooColIndexTestHostPtr = (int *) malloc(nnz_test * sizeof(cooColIndexTestHostPtr[0]));

  float* cooValHostTestPtr = (float *) malloc(nnz_test * sizeof(cooValHostTestPtr[0]));


  struct timeval tv0;
  gettimeofday(&tv0, NULL);


  loadCooSparseMatrixBin( (DATA_DIR + "/R_test_coo.data.bin").c_str(), (DATA_DIR + "/R_test_coo.row.bin").c_str(),
  (DATA_DIR + "/R_test_coo.col.bin").c_str(),cooValHostTestPtr, cooRowIndexTestHostPtr, cooColIndexTestHostPtr, nnz_test);

  cout<<"Reached here";
  printf("*******start loading training and testing sets to host.\n");

  loadCSRSparseMatrixBin( (DATA_DIR + "/R_train_csr.data.bin").c_str(), (DATA_DIR + "/R_train_csr.indptr.bin").c_str(),
  (DATA_DIR + "/R_train_csr.indices.bin").c_str(),csrValHostPtr, csrRowIndexHostPtr, csrColIndexHostPtr, m, nnz);

  loadCSCSparseMatrixBin( (DATA_DIR + "/R_train_csc.data.bin").c_str(), (DATA_DIR + "/R_train_csc.indices.bin").c_str(),
  (DATA_DIR +"/R_train_csc.indptr.bin").c_str(), cscValHostPtr, cscRowIndexHostPtr, cscColIndexHostPtr, n, nnz);

  loadCooSparseMatrixRowPtrBin( (DATA_DIR + "/R_train_coo.row.bin").c_str(), cooRowIndexHostPtr, nnz);

  double t0 = seconds();

  cout<<"Reached here";

  doALS(csrRowIndexHostPtr, csrColIndexHostPtr, csrValHostPtr,
			cscRowIndexHostPtr, cscColIndexHostPtr, cscValHostPtr,
			cooRowIndexHostPtr, thetaTHost, XTHost,
			cooRowIndexTestHostPtr, cooColIndexTestHostPtr, cooValHostTestPtr,
			m, n, f, nnz, nnz_test, lambda,
			ITERS, X_BATCH, THETA_BATCH,  0);

	printf("\ndoALS takes seconds: %.3f for F = %d\n", seconds() - t0, f);

  hipHostFree(csrRowIndexHostPtr);
	hipHostFree(csrColIndexHostPtr);
	hipHostFree(csrValHostPtr);
	hipHostFree(cscValHostPtr);
	hipHostFree(cscRowIndexHostPtr);
	hipHostFree(cscColIndexHostPtr);
	hipHostFree(cooRowIndexHostPtr);
	hipHostFree(XTHost);
	hipHostFree(thetaTHost);
	cudacall(hipDeviceReset());
	printf("\nALS Done.\n");
	return 0;
}
