
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <stdio.h>
using namespace std;

#define TILE_WIDTH 16
#define TILE_DIM 16
#define BLOCK_ROWS 16

// Compute C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
             int numARows, int numAColumns,
             int numBRows, int numBColumns,
             int numCRows, int numCColumns) {

    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * TILE_WIDTH + ty,
       Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;

    for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
       if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
          ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
       else
          ds_M[ty][tx] = 0;
       if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
          ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
       else
          ds_N[ty][tx] = 0;

       __syncthreads();
       for (int k = 0; k < TILE_WIDTH; ++k)
          Pvalue += ds_M[ty][k] * ds_N[k][tx];
       __syncthreads();
    }
    if (Row < numCRows && Col < numCColumns)
       C[Row*numCColumns+Col] = Pvalue;
}

__global__ void
transpose(float *odata, float *idata, int rows, int cols)
{
  // __shared__ float tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  // int width = gridDim.x * TILE_DIM;
  int width = cols;

  // for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  // {
  //   if(y+j<rows && x<cols)
  //     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];
  //   else
  //     tile[threadIdx.y+j][threadIdx.x] = 0.0;
  // }


  // __syncthreads();

  // x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  // y = blockIdx.x * TILE_DIM + threadIdx.y;

  // for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  // {
  //   if(y+j<rows && x<cols)
  //   { odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
  //     printf("%f\n", odata[(y+j)*width + x]);
  //   }
  // }
  if(x<cols && y<rows)
  {  odata[x*rows + y] = idata[y*cols + x];
  }
}
__global__
void elewisemulti(float *A, float *B, float *C, int rows, int cols)
{
  int tx = threadIdx.x, ty = threadIdx.y,
      bx = blockIdx.x, by = blockIdx.y;
  int row = by*blockDim.y+ty, col=blockDim.x*bx+tx;

  if(row<rows && col<cols)
    C[row*cols + col] = A[row*cols + col]*B[row*cols + col];
}

__global__
void elewisediv(float *A, float *B, float *C, int rows, int cols)
{
  int tx = threadIdx.x, ty = threadIdx.y,
      bx = blockIdx.x, by = blockIdx.y;
  int row = by*blockDim.y+ty, col=blockDim.x*bx+tx;

  if(row<rows && col<cols)
    C[row*cols + col] = A[row*cols + col]/B[row*cols + col];
}

int main()
{
  int m,n;
  ifstream dataset("dataset.txt");
  float *Ahost,*Bhost,*Chost,*Bhost1,*Chost1;
  float *A,*B,*C;
  //cout << "Enter dimensions of the matrix : ";
  //cin>>m>>n;
  m = 943;
  n = 1682;
  cout << "Enter k value : ";
  int k;
  cin>>k;
  int epochs;
  cout<<"Enter epochs for training : ";
  cin>>epochs;

  Ahost = new float[m*n];
  Bhost = new float[m*k];
  Bhost1 = new float[m*k];
  Chost = new float[k*n];
  Chost1 = new float[k*n];

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  srand(time(NULL));

  memset(Ahost,0,m*n*sizeof(float));

  for(int i=0;i<100000;++i){
  int temp;
  dataset >> temp;
  int x,y;
  x = --temp;
  dataset >> temp;
  y = --temp;
  dataset >> temp;
  Ahost[x*n + y] = temp;
  }

  for(int i=0;i<m;++i){
    for(int j=0;j<k;++j){
      Bhost[i*k + j] = rand()%2+0.1;
      Bhost1[i*k + j] = Bhost[i*k + j];
    }
  }

  //randomising the output array C
  for(int i=0;i<k;++i){
    for(int j=0;j<n;++j){
      Chost[i*n + j] = rand()%2+0.1;
      Chost1[i*n + j] = Chost[i*n + j];
    }
  }

  hipMalloc((void **)&A, sizeof(float) * m*n);
  hipMalloc((void **)&B, sizeof(float) * m*k);
  hipMalloc((void **)&C, sizeof(float) * k*n);

  hipMemcpy(A, Ahost, sizeof(float) * m*n, hipMemcpyHostToDevice);
  hipMemcpy(B, Bhost, sizeof(float) * m*k, hipMemcpyHostToDevice);
  hipMemcpy(C, Chost, sizeof(float) * k*n, hipMemcpyHostToDevice);


  float *ACtranspose = new float[m*k];
  float *Ctranspose = new float[k*n];
  float *BACtranspose = new float[m*k];
  float *CCtranspose = new float[k*k];
  float *BCCtranspose = new float[m*k];
  hipMalloc((void **)&ACtranspose, sizeof(float) * m*k);
  hipMalloc((void **)&Ctranspose, sizeof(float) * k*n);
  hipMalloc((void **)&CCtranspose, sizeof(float) * k*k);
  hipMalloc((void **)&BCCtranspose, sizeof(float) * m*k);
  hipMalloc((void **)&BACtranspose, sizeof(float) * m*k);

  dim3 gridSize1((n-1)/16 + 1,(k-1)/16 + 1,1);
  
  dim3 blockSize1(16, 16, 1);
  
  dim3 blockSize2(TILE_WIDTH, TILE_WIDTH, 1);


  dim3 gridSize2((k-1)/TILE_WIDTH + 1,(m-1)/TILE_WIDTH + 1,1);

  dim3 gridSize3((k-1)/TILE_WIDTH + 1,(k-1)/TILE_WIDTH + 1,1);


  float *BtransposeA = new float[k*n];
  float *Btranspose = new float[k*m];
  float *CBtransposeA = new float[k*n];
  float *BtransposeB = new float[k*k];
  float *BtransposeBC = new float[k*n];
  hipMalloc((void **)&BtransposeA, sizeof(float) * k*n);
  hipMalloc((void **)&Btranspose, sizeof(float) * k*m);
  hipMalloc((void **)&CBtransposeA, sizeof(float) * k*n);
  hipMalloc((void **)&BtransposeB, sizeof(float) * k*k);
  hipMalloc((void **)&BtransposeBC, sizeof(float) * k*n);

  dim3 gridSize4((k-1)/16 + 1,(m-1)/16 + 1,1);

  dim3 gridSize5((n-1)/TILE_WIDTH + 1,(k-1)/TILE_WIDTH + 1,1);

  dim3 gridSize6((k-1)/TILE_WIDTH + 1,(k-1)/TILE_WIDTH + 1,1);

  float *temp1 = new float[m*k];

  for(int i=0;i<epochs;i++)
  {

    transpose<<<gridSize1, blockSize1>>>(Ctranspose, C, k, n);
    hipDeviceSynchronize();



    matrixMultiply<<<gridSize2, blockSize2>>>(A, Ctranspose, ACtranspose, m, n, n, k, m, k);
    hipDeviceSynchronize();

    

    elewisemulti<<<gridSize2, blockSize2>>>(B, ACtranspose, BACtranspose, m, k);
    hipDeviceSynchronize();

    matrixMultiply<<<gridSize3, blockSize2>>>(C, Ctranspose, CCtranspose, k, n, n, k, k, k);
    hipDeviceSynchronize();

    matrixMultiply<<<gridSize2, blockSize2>>>(B, CCtranspose, BCCtranspose, m, k, k, k, m, k);
    hipDeviceSynchronize();
    elewisediv<<<gridSize2, blockSize2>>>(BACtranspose, BCCtranspose, B, m, k);
    hipDeviceSynchronize();

    

    transpose<<<gridSize4, blockSize1>>>(Btranspose, B, m, k);
    hipDeviceSynchronize();
    matrixMultiply<<<gridSize5, blockSize2>>>(Btranspose, A, BtransposeA, k, m, m, n, k, n);
    hipDeviceSynchronize();
    elewisemulti<<<gridSize5, blockSize2>>>(C, BtransposeA, CBtransposeA, k, n);
    hipDeviceSynchronize();

    matrixMultiply<<<gridSize6, blockSize2>>>(Btranspose, B, BtransposeB, k, m, m, k, k, k);
    hipDeviceSynchronize();
    matrixMultiply<<<gridSize2, blockSize2>>>(BtransposeB, C, BtransposeBC, k, k, k, n, k, n);
    hipDeviceSynchronize();
    elewisediv<<<gridSize2, blockSize2>>>(CBtransposeA, BtransposeBC, C, k, n);
    hipDeviceSynchronize();
  }

  hipMemcpy(Bhost, B, sizeof(float) * m*k, hipMemcpyDeviceToHost);
  hipMemcpy(Chost, C, sizeof(float) * k*n, hipMemcpyDeviceToHost);

  float temp[m][n];

  for(int i=0;i<m;++i){
    for(int j=0;j<n;++j){
      temp[i][j] = 0;
      for(int l=0;l<k;++l){
        temp[i][j]+= Bhost[i*k+l]*Chost[l*n+j];
      }
      // cout<<temp[i][j]<< " ";
    }
    // cout<<endl;
  }
  float sumError = 0;
  for(int i=0;i<m;++i){
    for(int j=0;j<n;++j){
      float errorMat  = Ahost[i*n+j] - temp[i][j];
      sumError += errorMat*errorMat;
    }
  }

  cout<<"RMS error : "<<sqrt(sumError/(m*n))<<endl;

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(ACtranspose);
  hipFree(Ctranspose);
  hipFree(CCtranspose);
  hipFree(BCCtranspose);
  hipFree(BACtranspose);
  hipFree(BtransposeA);
  hipFree(Btranspose);
  hipFree(CBtransposeA);
  hipFree(BtransposeB);
  hipFree(BtransposeBC);

  return 0;
}
