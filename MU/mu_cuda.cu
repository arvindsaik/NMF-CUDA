#include "hip/hip_runtime.h"
#include <bits/stdc++.h>

using namespace std;

#define TILE_WIDTH 16
#define TILE_DIM 32
#define BLOCK_ROWS 8

// Compute C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
  		       int numARows, int numAColumns,
			       int numBRows, int numBColumns,
			       int numCRows, int numCColumns) {

    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * TILE_WIDTH + ty,
       Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;

    for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
       if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
          ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
       else
          ds_M[ty][tx] = 0;
       if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
          ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
       else
          ds_N[ty][tx] = 0;

       __syncthreads();
       for (int k = 0; k < TILE_WIDTH; ++k)
          Pvalue += ds_M[ty][k] * ds_N[k][tx];
       __syncthreads();
    }
    if (Row < numCRows && Col < numCColumns)
       C[Row*numCColumns+Col] = Pvalue;
}

__global__ void transpose(float *odata, const float *idata, int rows, int cols)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  // int width = gridDim.x * TILE_DIM;
  int width = cols;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  {
    if(y+j<rows && x<cols)
      tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];
    else
      tile[threadIdx.y+j][threadIdx.x] = 0.0;
  }


  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  {
    if(y+j<rows && x<cols)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
  }
}

int main()
{
  int m,n;
  ifstream dataset("dataset.txt");
  float *Ahost,*Bhost,*Chost,*Bhost1,*Chost1;
  float *A,*B,*C;
  //cout << "Enter dimensions of the matrix : ";
  //cin>>m>>n;
  m = 943;
  n = 1682;
  cout << "Enter k value : ";
  int k;
  cin>>k;
  int epochs;
  cout<<"Enter epochs for training : ";
  cin>>epochs;

  Ahost = new float[m*n];
  Bhost = new float[m*k];
  Bhost1 = new float[m*k];
  Chost = new float[k*n];
  Chost1 = new float[k*n];

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  srand(time(NULL));

  memset(Ahost,0,m*n*sizeof(float));

  for(int i=0;i<100000;++i){
	int temp;
	dataset >> temp;
	int x,y;
	x = --temp;
	dataset >> temp;
	y = --temp;
	dataset >> temp;
	Ahost[x*n + y] = temp;
  }

  for(int i=0;i<m;++i){
    for(int j=0;j<k;++j){
      Bhost[i*k + j] = rand()%2;
      Bhost1[i*k + j] = Bhost[i*k + j];
    }
  }

  //randomising the output array C
  for(int i=0;i<k;++i){
    for(int j=0;j<n;++j){
      Chost[i*n + j] = rand()%2;
      Chost1[i*n + j] = Chost[i*n + j];
    }
  }

  hipMalloc((void **)&A, sizeof(float) * m*n);
  hipMalloc((void **)&B, sizeof(float) * m*k);
  hipMalloc((void **)&C, sizeof(float) * k*n);

  hipMemcpy(A, Ahost, sizeof(float) * m*n, hipMemcpyHostToDevice);
  hipMemcpy(B, Bhost, sizeof(float) * m*k, hipMemcpyHostToDevice);
  hipMemcpy(C, Chost, sizeof(float) * k*n, hipMemcpyHostToDevice);


  float *ACtranspose = new float[m*k];
  float *Ctranspose = new float[k*n];
  hipMalloc((void **)&ACtranspose, sizeof(float) * m*k);
  hipMalloc((void **)&Ctranspose, sizeof(float) * k*n);

  dim3 gridSize1((k-1)/TILE_DIM + 1,(n-1)/TILE_DIM + 1,1);
  dim3 blockSize1(TILE_DIM, BLOCK_ROWS, 1);

  dim3 gridSize2((m-1)/TILE_WIDTH + 1,(k-1)/TILE_WIDTH + 1,1);
  dim3 blockSize2(TILE_WIDTH, TILE_WIDTH, 1);

  for(int i=0;i<epochs;i++)
  {

    transpose<<<gridSize, blockSize>>>(Ctranspose, C, k, n);
    matrixMultiply<<<gridSize, blockSize>>>(A, Ctranspose, ACtranspose, m, n, n, k, m, k);
  }

  hipFree(A);
  hipFree(B);
  hipFree(C);


}
