#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <stdio.h>
using namespace std;

#define TILE_WIDTH 16
#define TILE_DIM 16
#define BLOCK_ROWS 16

// Compute C = A * B
_global_ void matrixMultiply(float * A, float * B, float * C,
             int numARows, int numAColumns,
             int numBRows, int numBColumns,
             int numCRows, int numCColumns) {

    _shared_ float ds_M[TILE_WIDTH][TILE_WIDTH];
    _shared_ float ds_N[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * TILE_WIDTH + ty,
       Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;

    for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
       if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
          ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
       else
          ds_M[ty][tx] = 0;
       if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
          ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
       else
          ds_N[ty][tx] = 0;

       __syncthreads();
       for (int k = 0; k < TILE_WIDTH; ++k)
          Pvalue += ds_M[ty][k] * ds_N[k][tx];
       __syncthreads();
    }
    if (Row < numCRows && Col < numCColumns)
       C[Row*numCColumns+Col] = Pvalue;
}

_global_ void
transpose(float *odata, float *idata, int rows, int cols)
{
  _shared_ float tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;

  // int width = cols;
  //
  // // for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  // // {
  //   if(y<rows && x<cols){
  //     tile[threadIdx.y][threadIdx.x] = idata[(y)*width + x];
  //
  //      // printf("%f   ",  tile[threadIdx.y][threadIdx.x]);
  //   }
  //
  // __syncthreads();
  //
  //
  // x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  // y = blockIdx.x * TILE_DIM + threadIdx.y;
  //
  // // for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
  // // {
  //    if(y<cols && x<rows)
  //   { odata[(y)*width + x] = tile[threadIdx.x][threadIdx.y];
  //     // printf("%f\n", tile[threadIdx.x][threadIdx.y]);
  //   }
  // // }
  if(x<cols && y<rows)
  {  odata[x*rows + y] = idata[y*cols + x];
  }
}
_global_
void elewisemulti(float *A, float *B, float *C, int rows, int cols)
{
  int tx = threadIdx.x, ty = threadIdx.y,
      bx = blockIdx.x, by = blockIdx.y;
  int row = by*blockDim.y+ty, col=blockDim.x*bx+tx;

  if(row<rows && col<cols)
    C[row*cols + col] = A[row*cols + col]*B[row*cols + col];
}

_global_
void elewisediv(float *A, float *B, float *C, int rows, int cols)
{
  int tx = threadIdx.x, ty = threadIdx.y,
      bx = blockIdx.x, by = blockIdx.y;
  int row = by*blockDim.y+ty, col=blockDim.x*bx+tx;

  if(row<rows && col<cols)
    C[row*cols + col] = A[row*cols + col]/B[row*cols + col];
}

int main()
{
  int m,n;
  ifstream dataset("dataset.txt");
  float *Ahost,*Bhost,*Chost,*Bhost1,*Chost1;
  float *A,*B,*C;
  //cout << "Enter dimensions of the matrix : ";
  //cin>>m>>n;
  m = 943;
  n = 1682;
  // cout << "Enter k value : ";
  int k;
  // cin>>k;
  k=10;
  int epochs;
  // cout<<"Enter epochs for training : ";
  // cin>>epochs;
  epochs = 10;

  Ahost = new float[m*n];
  Bhost = new float[m*k];
  Bhost1 = new float[m*k];
  Chost = new float[k*n];
  Chost1 = new float[k*n];

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  srand(time(NULL));

  memset(Ahost,0,m*n*sizeof(float));

  for(int i=0;i<100000;++i){
  int temp;
  dataset >> temp;
  int x,y;
  x = --temp;
  dataset >> temp;
  y = --temp;
  dataset >> temp;
  Ahost[x*n + y] = temp;
  }

  for(int i=0;i<m;++i){
    for(int j=0;j<k;++j){
      Bhost[i*k + j] = ((float) rand() / (RAND_MAX));
      Bhost1[i*k + j] = Bhost[i*k + j];
    }
  }

  //randomising the output array C
  for(int i=0;i<k;++i){
    for(int j=0;j<n;++j){
      Chost[i*n + j] = ((float) rand() / (RAND_MAX));
      Chost1[i*n + j] = Chost[i*n + j];
    }
  }

  hipMalloc((void **)&A, sizeof(float) * m*n);
  hipMalloc((void **)&B, sizeof(float) * m*k);
  hipMalloc((void **)&C, sizeof(float) * k*n);

  hipMemcpy(A, Ahost, sizeof(float) * m*n, hipMemcpyHostToDevice);
  hipMemcpy(B, Bhost, sizeof(float) * m*k, hipMemcpyHostToDevice);
  hipMemcpy(C, Chost, sizeof(float) * k*n, hipMemcpyHostToDevice);


  float *ACtranspose = new float[m*k];
  float *Ctranspose = new float[k*n];
  float *BACtranspose = new float[m*k];
  float *CCtranspose = new float[k*k];
  float *BCCtranspose = new float[m*k];
  hipMalloc((void **)&ACtranspose, sizeof(float) * m*k);
  hipMalloc((void **)&Ctranspose, sizeof(float) * k*n);
  hipMalloc((void **)&CCtranspose, sizeof(float) * k*k);
  hipMalloc((void **)&BCCtranspose, sizeof(float) * m*k);
  hipMalloc((void **)&BACtranspose, sizeof(float) * m*k);

  dim3 gridSize1((n-1)/16 + 1,(k-1)/16 + 1,1);

  dim3 blockSize1(16, 16, 1);

  dim3 blockSize2(TILE_WIDTH, TILE_WIDTH, 1);


  dim3 gridSize2((k-1)/TILE_WIDTH + 1,(m-1)/TILE_WIDTH + 1,1);

  dim3 gridSize3((k-1)/TILE_WIDTH + 1,(k-1)/TILE_WIDTH + 1,1);


  float *BtransposeA = new float[k*n];
  float *Btranspose = new float[k*m];
  float *CBtransposeA = new float[k*n];
  float *BtransposeB = new float[k*k];
  float *BtransposeBC = new float[k*n];
  hipMalloc((void **)&BtransposeA, sizeof(float) * k*n);
  hipMalloc((void **)&Btranspose, sizeof(float) * k*m);
  hipMalloc((void **)&CBtransposeA, sizeof(float) * k*n);
  hipMalloc((void **)&BtransposeB, sizeof(float) * k*k);
  hipMalloc((void **)&BtransposeBC, sizeof(float) * k*n);

  dim3 gridSize4((k-1)/16 + 1,(m-1)/16 + 1,1);

  dim3 gridSize5((n-1)/TILE_WIDTH + 1,(k-1)/TILE_WIDTH + 1,1);

  dim3 gridSize6((k-1)/TILE_WIDTH + 1,(k-1)/TILE_WIDTH + 1,1);

  float *temp1 = new float[n*k];

  hipEventRecord(start);
  for(int i=0;i<epochs;i++)
  {

    transpose<<<gridSize1, blockSize1>>>(Ctranspose, C, k, n);

    hipMemcpy(temp1, Ctranspose, sizeof(float) * n*k, hipMemcpyDeviceToHost);
    // for(int p=0;p<n*k;p++)
    //   cout<<temp1[p]<<" ";

    matrixMultiply<<<gridSize2, blockSize2>>>(A, Ctranspose, ACtranspose, m, n, n, k, m, k);



    elewisemulti<<<gridSize2, blockSize2>>>(B, ACtranspose, BACtranspose, m, k);



    matrixMultiply<<<gridSize3, blockSize2>>>(C, Ctranspose, CCtranspose, k, n, n, k, k, k);



    matrixMultiply<<<gridSize2, blockSize2>>>(B, CCtranspose, BCCtranspose, m, k, k, k, m, k);



    elewisediv<<<gridSize2, blockSize2>>>(BACtranspose, BCCtranspose, B, m, k);



     transpose<<<gridSize4, blockSize1>>>(Btranspose, B, m, k);


    matrixMultiply<<<gridSize5, blockSize2>>>(Btranspose, A, BtransposeA, k, m, m, n, k, n);


    elewisemulti<<<gridSize5, blockSize2>>>(C, BtransposeA, CBtransposeA, k, n);

    matrixMultiply<<<gridSize6, blockSize2>>>(Btranspose, B, BtransposeB, k, m, m, k, k, k);

    matrixMultiply<<<gridSize5, blockSize2>>>(BtransposeB, C, BtransposeBC, k, k, k, n, k, n);

    elewisediv<<<gridSize5, blockSize2>>>(CBtransposeA, BtransposeBC, C, k, n);

  }

    hipEventRecord(stop);

  hipMemcpy(Bhost, B, sizeof(float) * m*k, hipMemcpyDeviceToHost);
  hipMemcpy(Chost, C, sizeof(float) * k*n, hipMemcpyDeviceToHost);
  hipEventSynchronize(stop);


      float milliseconds = 0;
      hipEventElapsedTime(&milliseconds, start, stop);

      cout<<"GPU kernel took "<<milliseconds/1000<<" seconds"<<endl;

  float temp[m][n];

  for(int i=0;i<m;++i){
    for(int j=0;j<n;++j){
      temp[i][j] = 0;
      for(int l=0;l<k;++l){
        temp[i][j]+= Bhost[i*k+l]*Chost[l*n+j];
      }
      // cout<<temp[i][j]<< " ";
    }
    // cout<<endl;
  }
  float sumError = 0;
  for(int i=0;i<m;++i){
    for(int j=0;j<n;++j){
      float errorMat  = Ahost[i*n+j] - temp[i][j];
      sumError += errorMat*errorMat;
    }
  }

  cout<<"RMS error : "<<sqrt(sumError/(m*n))<<endl;

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(ACtranspose);
  hipFree(Ctranspose);
  hipFree(CCtranspose);
  hipFree(BCCtranspose);
  hipFree(BACtranspose);
  hipFree(BtransposeA);
  hipFree(Btranspose);
  hipFree(CBtransposeA);
  hipFree(BtransposeB);
  hipFree(BtransposeBC);

  return 0;
}
