
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

void sgd_CPU(float *A,float *B,float *C,int epochs,float lamda,float alpha,int m,int n,int k){
    for(int i=0;i<epochs;++i){
      for(int x = 0;x<m;++x){
        for(int y = 0;y<n;++y){
          float error = 0;
          float temp = 0;
          for(int iter = 0;iter < k;++iter){
            temp += B[x*k + iter]*C[iter*n + y];
          }
          error = A[x*n+y] - temp;
          for(int iter = 0;iter < k;++iter){
            B[x*k + iter] = B[x*k + iter] + alpha*((error * C[iter*n + y]) - lamda*(B[x*k + iter]));
            C[iter*n + y] = C[iter*n + y] + alpha*((error * B[x*k + iter]) - lamda*(C[iter*n + y]));
          }
        }
      }
    }
}

__global__ void sgd_kernel(float *A,float *B,float *C,int epochs,float lamda,float alpha,int m,int n,int k){
	for(int eps=0;eps<epochs;++eps){ 
		int y = blockIdx.x * blockDim.x + threadIdx.x;
		int x = blockIdx.y * blockDim.y + threadIdx.y;
		if(y<k && x<m){
			float error = 0;
			float gradient = 0;
			for(int j=0;j<n;++j){
				float temp = 0;
				for(int iter=0;iter<k;++iter){
					temp += B[x*k + iter]*C[iter*n + j];
				}
				error = A[x*n + j] - temp;
				gradient += error*C[y*n + j];
			}
			__syncthreads();
			B[x*k + y] += alpha*(gradient - lamda*(B[x*k + y]));
		}
		y = blockIdx.x * blockDim.x + threadIdx.x;
		x = blockIdx.y * blockDim.y + threadIdx.y;
		if(y<n && x<k){
			float error = 0;
			float gradient = 0;
			for(int i=0;i<m;++i){
				float temp = 0;
				for(int iter=0;iter<k;++iter){
					temp += B[i*k + iter]*C[iter*n + y];
				}
				error = A[i*n + y] - temp;
				gradient += error*B[i*k + x];
			}
			__syncthreads();
			C[x*n + y] += alpha*(gradient - lamda*(C[x*n + y]));
		}
		__syncthreads();
	}
}

int main(){
  int m,n;
  ifstream dataset("dataset.txt");
  float *Ahost,*Bhost,*Chost,*Bhost1,*Chost1;
  float *A,*B,*C;
  //cout << "Enter dimensions of the matrix : ";
  //cin>>m>>n;
  m = 943;
  n = 1682;
  cout << "Enter k value : ";
  int k;
  cin>>k;
  int epochs;
  cout<<"Enter epochs for training : ";
  cin>>epochs;
  float alpha;
  cout<<"Enter alpha for training : ";
  cin>>alpha;
  float lamda;
  cout<<"Enter lamda (regularisation variable) for training : ";
  cin>>lamda;
  Ahost = new float[m*n];
  Bhost = new float[m*k];
  Bhost1 = new float[m*k];
  Chost = new float[k*n];
  Chost1 = new float[k*n];
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  srand(time(NULL));

  memset(Ahost,0,sizeof(Ahost));

  for(int i=0;i<100000;++i){
	int temp;
	dataset >> temp;
	int x,y;
	x = --temp;
	dataset >> temp;
	y = --temp;
	dataset >> temp;
	Ahost[x*n + y] = temp;
  }
/*
  //randomising the input array A
  for(int i=0;i<m;++i){
    for(int j=0;j<n;++j){
      Ahost[i*n + j] = rand()%5 + 1.03;
    }
  }
*/

  // randomising the output array B
  for(int i=0;i<m;++i){
    for(int j=0;j<k;++j){
      Bhost[i*k + j] = rand()%2;
      Bhost1[i*k + j] = Bhost[i*k + j];
    }
  }

  //randomising the output array C
  for(int i=0;i<k;++i){
    for(int j=0;j<n;++j){
      Chost[i*n + j] = rand()%2;
      Chost1[i*n + j] = Chost[i*n + j];
    }
  }

  // cout<<" A : "<<endl;
  
  // for(int i=0;i<m;++i){
  //   for(int j=0;j<n;++j){
  //     cout<<Ahost[i*n+j]<<" ";
  //   }
  //   cout<<endl;
  // }

  // cout<<" B : \n";
  //
  // for(int i=0;i<m;++i){
  //   for(int j=0;j<k;++j){
  //     cout<<Bhost[i*k+j]<< " ";
  //   }
  //   cout<<endl;
  // }

  hipMalloc((void **)&A, sizeof(float) * m*n);
  hipMalloc((void **)&B, sizeof(float) * m*k);
  hipMalloc((void **)&C, sizeof(float) * k*n);

  hipMemcpy(A, Ahost, sizeof(float) * m*n, hipMemcpyHostToDevice);
  hipMemcpy(B, Bhost, sizeof(float) * m*k, hipMemcpyHostToDevice);
  hipMemcpy(C, Chost, sizeof(float) * k*n, hipMemcpyHostToDevice);


  dim3 gridSize((n-1)/32 + 1,(m-1)/32 + 1,1);
  dim3 blockSize(32,32,1);

  hipEventRecord(start);

  sgd_kernel<<<gridSize,blockSize>>>(A,B,C,epochs,lamda,alpha,m,n,k);

  hipEventRecord(stop);


  hipMemcpy(Bhost, B, sizeof(float) * m*k, hipMemcpyDeviceToHost);
  hipMemcpy(Chost, C, sizeof(float) * k*n, hipMemcpyDeviceToHost);

  // cout<<" Product : \n";
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  cout<<"GPU kernel took "<<milliseconds/1000<<" seconds"<<endl;

  float temp[m][n];

  for(int i=0;i<m;++i){
    for(int j=0;j<n;++j){
      temp[i][j] = 0;
      for(int l=0;l<k;++l){
        temp[i][j]+= Bhost[i*k+l]*Chost[l*n+j];
      }
      // cout<<temp[i][j]<< " ";
    }
    // cout<<endl;
  }
  float sumError = 0;
  for(int i=0;i<m;++i){
    for(int j=0;j<n;++j){
      float errorMat  = Ahost[i*n+j] - temp[i][j];
      sumError += errorMat*errorMat;
    }
  }

  cout<<"RMS error : "<<sqrt(sumError/(m*n))<<endl;

  double s = clock();
  sgd_CPU(Ahost,Bhost1,Chost1,epochs,lamda,alpha,m,n,k);
  double e = clock();

  cout<<"CPU implementation took "<< (e-s)/CLOCKS_PER_SEC<<" seconds\n";

  for(int i=0;i<m;++i){
    for(int j=0;j<n;++j){
      temp[i][j] = 0;
      for(int l=0;l<k;++l){
        temp[i][j]+= Bhost1[i*k+l]*Chost1[l*n+j];
      }
      // cout<<temp[i][j]<< " ";
    }
    // cout<<endl;
  }
  sumError = 0;
  for(int i=0;i<m;++i){
    for(int j=0;j<n;++j){
      float errorMat  = Ahost[i*n+j] - temp[i][j];
      sumError += errorMat*errorMat;
    }
  }

  cout<<"RMS error : "<<sqrt(sumError/(m*n))<<endl;

  return 0;
}
